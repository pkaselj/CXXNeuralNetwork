#include "hip/hip_runtime.h"
#include "FullyConnectedLayer_CUDA.cuh"

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

#include <>
#include <memory>
#include <cstdlib>

static __global__ void PerformMatrixMultiplicationInCUDA(
	int X, int Y,
	double* CurrentLayerNeurons,
	double* Weights,
	double* Biases,
	double* NextLayerNeurons
)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;

	printf("i = %d, j = %d, N = %lf, P = %lf, W = %lf\n", i, j, NextLayerNeurons[i], CurrentLayerNeurons[i], Weights[i * Y + j]);

	NextLayerNeurons[i] += CurrentLayerNeurons[i] * Weights[i * Y + j];

	//printf("%lf\n", NextLayerNeurons[j]);

	//NextLayerNeurons[i] = sum + Biases[i];

}

void FullyConnectedLayer_CUDA::PerformMatrixMultiplication(
	int X, int Y,
	const Vector1D& CurrentLayerNeurons,
	const Vector2D& Weights,
	const Vector1D& Biases,
	Vector1D& NextLayerNeurons
)
{
	// Launching the kernel
	dim3 threadsPerBlock(X, Y);
	dim3 numBlocks(1);

	double* p_current_layer;
	double* p_weights;
	double* p_biases;
	double* p_next_layer;


	int size_of_array = X * Y * sizeof(double);
	hipMalloc((void**)&p_weights, size_of_array);
	for (int i = 0; i < X; i++)
	{
		hipMemcpy(p_weights + Y * i, Weights[i].data(), Y * sizeof(double), hipMemcpyHostToDevice);
	}

	hipMalloc((void**)&p_current_layer, CurrentLayerNeurons.size() * sizeof(double));
	hipMemcpy(p_current_layer, CurrentLayerNeurons.data(), CurrentLayerNeurons.size() * sizeof(double), hipMemcpyHostToDevice);

	hipMalloc((void**)&p_biases, Biases.size() * sizeof(double));
	hipMemcpy(p_biases, Biases.data(), Biases.size() * sizeof(double), hipMemcpyHostToDevice);

	hipMalloc((void**)&p_next_layer, NextLayerNeurons.size() * sizeof(double));
	hipMemcpy(p_next_layer, NextLayerNeurons.data(), NextLayerNeurons.size() * sizeof(double), hipMemcpyHostToDevice);


	PerformMatrixMultiplicationInCUDA <<<numBlocks, threadsPerBlock >>>(X, Y, p_current_layer, p_weights, p_biases, p_next_layer);

	hipMemcpy(NextLayerNeurons.data(), p_next_layer, NextLayerNeurons.size(), hipMemcpyDeviceToHost);

	hipFree(p_current_layer);
	hipFree(p_biases);
	hipFree(p_next_layer);
	hipFree(p_weights);


}




#include "hip/hip_runtime.h"
#include "FullyConnectedLayer_CUDA.cuh"

#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

#include <>
#include <memory>
#include <cstdlib>


#include <chrono>
#include <iostream>
static __global__ void PerformMatrixMultiplicationInCUDA(
	int X, int Y,
	double* CurrentLayerNeurons,
	double* Weights,
	double* Biases,
	double* NextLayerNeurons
)
{
	int col = blockIdx.x * blockDim.x + threadIdx.x;

	// Guard against extra threads
	if (col < Y) {
		// Calculate dot product of CurrentLayer vector and Weight's "col" column
		for (int i = 0; i < X; i++)
		{
#if defined(_DEBUG)
			printf("col = %d, i = %d, N[%d] = C[%d] (%lf) * W[%d] (%lf) = (%lf)\n",
				col, i,
				col,
				i, CurrentLayerNeurons[i],
				i * Y + col, Weights[i * Y + col],
				CurrentLayerNeurons[i] * Weights[i * Y + col]
			);
#endif

			NextLayerNeurons[col] += CurrentLayerNeurons[i] * Weights[i * Y + col];
		}
		// Add Bias
		NextLayerNeurons[col] += Biases[col];

#if defined(_DEBUG)
		printf("col = %d, N = %lf\n", col, NextLayerNeurons[col]);
#endif
	}
	

}

void FullyConnectedLayer_CUDA::PerformMatrixMultiplication(
	int X, int Y,
	const Vector1D& CurrentLayerNeurons,
	const Vector2D& Weights,
	const Vector1D& Biases,
	Vector1D& NextLayerNeurons
)
{
	double* p_current_layer;
	double* p_weights;
	double* p_biases;
	double* p_next_layer;


	// -- Copy Weights to Device

	int size_of_array = X * Y * sizeof(double);

	hipMalloc(
		(void**)&p_weights,	// Destination
		size_of_array		// Size in Bytes
	);

	for (int i = 0; i < X; i++)
	{
		hipMemcpy(
			p_weights + Y * i,		// Destination Row
			Weights[i].data(),		// Source Row
			Y * sizeof(double),		// Size in Bytes
			hipMemcpyHostToDevice	// Copy Direction
		);
	}

	// -- Copy Current Layer Neuron Values to Device

	hipMalloc(
		(void**)&p_current_layer,						// Destination
		CurrentLayerNeurons.size() * sizeof(double)		// Size in Bytes
	);

	hipMemcpy(
		(void*)p_current_layer,						   // Destination Row
		(const void*)CurrentLayerNeurons.data(),	   // Source Row
		CurrentLayerNeurons.size() * sizeof(double),   // Size in Bytes
		hipMemcpyHostToDevice						   // Copy Direction
	);

	hipMalloc(
		(void**)&p_biases,					   // Destination
		Biases.size() * sizeof(double)		   // Size in Bytes
	);

	// -- Copy Bias Values to Device

	hipMemcpy(
		(void*)p_biases,					  // Destination Row
		(const void*)Biases.data(),			  // Source Row
		Biases.size() * sizeof(double),		  // Size in Bytes
		hipMemcpyHostToDevice				  // Copy Direction
	);

	// -- Allocate Buffer for Results (Values of Next Layer)

	hipMalloc(
		(void**)&p_next_layer,						  // Destination
		NextLayerNeurons.size() * sizeof(double)	  // Size in Bytes
	);

	//hipMemcpy(
	//	(void*)p_next_layer,						   // Destination Row
	//	(const void*)NextLayerNeurons.data(),		   // Source Row
	//	NextLayerNeurons.size() * sizeof(double),	   // Size in Bytes
	//	hipMemcpyHostToDevice						   // Copy Direction
	//);


	// -- Perform Matrix Multiplication on GPU
	int threadsPerBlock = 16;
	int numBlocks = ceil((1.0f * Y) / threadsPerBlock);

	auto start = std::chrono::high_resolution_clock::now();

	PerformMatrixMultiplicationInCUDA <<<numBlocks, threadsPerBlock >>>(
		X, Y,
		p_current_layer,
		p_weights,
		p_biases,
		p_next_layer
	);

	/* PROCESSING NEURAL NETWORK INPUT */
	auto finish = std::chrono::high_resolution_clock::now();
	auto microseconds = std::chrono::duration_cast<std::chrono::microseconds>(finish - start);
	std::cout << microseconds.count() << " us\n";

	hipMemcpy(
		NextLayerNeurons.data(),					// Destination Row
		p_next_layer,								// Source Row
		NextLayerNeurons.size() * sizeof(double),	// Size in Bytes
		hipMemcpyDeviceToHost					    // Copy Direction
	);


	// -- Deallocate Device Buffers

	hipFree(p_current_layer);
	hipFree(p_biases);
	hipFree(p_next_layer);
	hipFree(p_weights);


}




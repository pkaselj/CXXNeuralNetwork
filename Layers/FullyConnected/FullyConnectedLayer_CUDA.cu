#include "hip/hip_runtime.h"
#include "FullyConnectedLayer_CUDA.cuh"

#include <crt\hip/device_functions.h>
#include <hip/hip_runtime.h>

#include <hip/hip_runtime_api.h>

#include <>


//Helper function to calculate size of vector of vectors
int calculateSize(const std::vector<std::vector<double>>& input) {
	int size = 0;
	for (int i = 0; i < input.size(); i++) {
		size += input[i].size() * sizeof(double);
	}
	return size;
}

//Function to copy std::vector<std::vector<double>> from host to device
extern "C" void copyVecToDevice(const std::vector<std::vector<double>> &input, double* d_input) {
	int size = calculateSize(input);
	hipMalloc((void**)&d_input, size);
	int offset = 0;
	for (int i = 0; i < input.size(); i++) {
		hipMemcpy(d_input + offset, input[i].data(), input[i].size() * sizeof(double), hipMemcpyHostToDevice);
		offset += input[i].size();
	}
}

static __global__ void PerformMatrixMultiplicationInCUDA(
	int X, int Y,
	double* CurrentLayerNeurons,
	double** Weights,
	double* Biases,
	double* NextLayerNeurons
)
{
	int i = blockIdx.x * blockDim.x + threadIdx.x;
	int j = blockIdx.y * blockDim.y + threadIdx.y;
	double sum = 0;
	for (int k = 0; k < X; k++)
	{
		sum += CurrentLayerNeurons[k] * Weights[k][j];
	}
	NextLayerNeurons[i] = sum + Biases[j];
}

void FullyConnectedLayer_CUDA::PerformMatrixMultiplication(
	int X, int Y,
	const Vector1D& CurrentLayerNeurons,
	const Vector2D& Weights,
	const Vector1D& Biases,
	Vector1D& NextLayerNeurons
)
{
	// Launching the kernel
	dim3 threadsPerBlock(X, Y);
	dim3 numBlocks(1);

	double* p_current_layer;
	double** p_weights; // 2D
	double* p_biases;
	double* p_next_layer;

	copyVecToDevice(Weights, *p_weights);

	hipMalloc((void**)&p_current_layer, CurrentLayerNeurons.size() * sizeof(double));
	hipMemcpy(p_current_layer, CurrentLayerNeurons.data(), CurrentLayerNeurons.size() * sizeof(double), hipMemcpyHostToDevice);

	hipMalloc((void**)&p_biases, Biases.size() * sizeof(double));
	hipMemcpy(p_biases, Biases.data(), Biases.size() * sizeof(double), hipMemcpyHostToDevice);

	hipMalloc((void**)&p_next_layer, NextLayerNeurons.size() * sizeof(double));
	hipMemcpy(p_next_layer, NextLayerNeurons.data(), NextLayerNeurons.size() * sizeof(double), hipMemcpyHostToDevice);


	PerformMatrixMultiplicationInCUDA <<<numBlocks, threadsPerBlock >>>(X, Y, p_current_layer, p_weights, p_biases, p_next_layer);

	hipFree(p_current_layer);
	hipFree(p_biases);
	hipFree(p_next_layer);

}



